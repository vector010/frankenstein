#include "hip/hip_runtime.h"
/* Copyright (C) 1883 Thomas Edison - All Rights Reserved
 * You may use, distribute and modify this code under the
 * terms of the GPLv3 license, which unfortunately won't be
 * written for another century.
 *
 * You should have received a copy of the LICENSE file with
 * this file.
 */

#include "frkhash_cuda_miner_kernel.h"

#include "frkhash_cuda_miner_kernel_globals.h"

#include "cuda_helper.h"

#define copy(dst, src, count)                                                                                          \
    for (int i = 0; i != count; ++i) {                                                                                 \
        (dst)[i] = (src)[i];                                                                                           \
    }

#include "keccak.cuh"

#include "hash_it.cuh"

__global__ void frkhash_search(Search_results* g_output, uint64_t start_nonce) {
    if (g_output->done)
        return;
    uint32_t const gid = blockIdx.x * blockDim.x + threadIdx.x;
    bool r = compute_hash(start_nonce + gid);
    if (threadIdx.x == 0)
        atomicInc((uint32_t*)&g_output->hashCount, 0xffffffff);
    if (r)
        return;
    uint32_t index = atomicInc((uint32_t*)&g_output->solCount, 0xffffffff);
    if (index >= MAX_SEARCH_RESULTS)
        return;
    g_output->gid[index] = gid;
    g_output->done = 1;
}

void run_frkhash_search(uint32_t gridSize, uint32_t blockSize, hipStream_t stream, Search_results* g_output,
                       uint64_t start_nonce) {
    frkhash_search<<<gridSize, blockSize, 0, stream>>>(g_output, start_nonce);
    CUDA_CALL(hipGetLastError());
}

/*
__global__ void frkhash_calculate_dag_item(uint32_t start) {
    uint32_t const node_index = start + blockIdx.x * blockDim.x + threadIdx.x;
    if (((node_index >> 1) & (~1)) >= d_dag_size)
        return;
    union {
        hash128_t dag_node;
        uint2 sha3_buf[25];
    };
    copy(dag_node.uint4s, d_light[node_index % d_light_size].uint4s, 4);
    dag_node.words[0] ^= node_index;
    SHA3_512(sha3_buf);

    const int thread_id = threadIdx.x & 3;

    for (uint32_t i = 0; i != FRKHASH_DATASET_PARENTS; ++i) {
        uint32_t parent_index = fnv(node_index ^ i, dag_node.words[i % NODE_WORDS]) % d_light_size;
        for (uint32_t t = 0; t < 4; t++) {
            uint32_t shuffle_index = SHFL(parent_index, t, 4);

            uint4 p4 = d_light[shuffle_index].uint4s[thread_id];
            for (int w = 0; w < 4; w++) {
                uint4 s4 = make_uint4(SHFL(p4.x, w, 4), SHFL(p4.y, w, 4), SHFL(p4.z, w, 4), SHFL(p4.w, w, 4));
                if (t == thread_id) {
                    dag_node.uint4s[w] = fnv4(dag_node.uint4s[w], s4);
                }
            }
        }
    }
    SHA3_512(sha3_buf);
    hash64_t* dag_nodes = (hash64_t*)d_dag;
    copy(dag_nodes[node_index].uint4s, dag_node.uint4s, 4);
}
*/
void set_header(hash32_t _header) { CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_header), &_header, sizeof(hash32_t))); }

void set_target(uint64_t _target) { CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_target), &_target, sizeof(uint64_t))); }
